
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <iostream>
#include <chrono>

using namespace std::chrono;

/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(cudaMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(cudaGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        std::cerr << "cuda error: " << hipGetErrorString(result) << std::endl;
        exit(1);
    }
}

__global__ void vectorAddKernel(float* A, float* B, float* Result) {
    // insert operation here
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    Result[i] = A[i] + B[i];
}

void vectorAddCuda(int n, float* a, float* b, float* result) {
    int threadBlockSize;

    if (n < 512)
        threadBlockSize = n;
    else
        threadBlockSize = 512;

    // allocate the vectors on the GPU
    float* deviceA = NULL;
    checkCudaCall(hipMalloc((void **) &deviceA, n * sizeof(float)));
    if (deviceA == NULL) {
        std::cout << "could not allocate memory!" << std::endl;
        return;
    }
    float* deviceB = NULL;
    checkCudaCall(hipMalloc((void **) &deviceB, n * sizeof(float)));
    if (deviceB == NULL) {
        checkCudaCall(hipFree(deviceA));
        std::cout << "could not allocate memory!" << std::endl;
        return;
    }
    float* deviceResult = NULL;
    checkCudaCall(hipMalloc((void **) &deviceResult, n * sizeof(float)));
    if (deviceResult == NULL) {
        checkCudaCall(hipFree(deviceA));
        checkCudaCall(hipFree(deviceB));
        std::cout << "could not allocate memory!" << std::endl;
        return;
    }

    high_resolution_clock::time_point t1 = high_resolution_clock::now();

    // copy the original vectors to the GPU
    checkCudaCall(hipMemcpy(deviceA, a, n*sizeof(float), hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(deviceB, b, n*sizeof(float), hipMemcpyHostToDevice));

    high_resolution_clock::time_point t2 = high_resolution_clock::now();

    // execute kernel
    vectorAddKernel<<<n/threadBlockSize, threadBlockSize>>>(deviceA, deviceB, deviceResult);
    hipDeviceSynchronize();

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    high_resolution_clock::time_point t3 = high_resolution_clock::now();

    // copy result back
    checkCudaCall(hipMemcpy(result, deviceResult, n * sizeof(float), hipMemcpyDeviceToHost));

    high_resolution_clock::time_point t4 = high_resolution_clock::now();

    checkCudaCall(hipFree(deviceA));
    checkCudaCall(hipFree(deviceB));
    checkCudaCall(hipFree(deviceResult));

    std::cout << "vector-add (H2D):    \t\t" << duration_cast<microseconds>(t2 - t1).count() << "us" << std::endl;
    std::cout << "vector-add (kernel): \t\t" << duration_cast<microseconds>(t3 - t2).count() << "us" << std::endl;
    std::cout << "vector-add (D2H):    \t\t" << duration_cast<microseconds>(t4 - t3).count() << "us" << std::endl;
}

void vectorAddSeq(int n, float* a, float* b, float* result) {
    int i;

    high_resolution_clock::time_point t1 = high_resolution_clock::now();

    for (i=0; i<n; i++) {
        result[i] = a[i]+b[i];
    }

    high_resolution_clock::time_point t2 = high_resolution_clock::now();

    std::cout << "vector-add (seq):    \t\t" << duration_cast<microseconds>(t2 - t1).count() << "us" << std::endl;
}

int main(int argc, char* argv[]) {
    // int n = 655360;
    int n = atoi(getenv("VECTOR_ADD_N"));
    float* a = new float[n];
    float* b = new float[n];
    float* result = new float[n];
    float* result_s = new float[n];

    if (argc > 1) n = atoi(argv[1]);

    std::cout << "Adding two vectors of " << n << " integer elements." << std::endl;

    // initialize the vectors.
    for(int i=0; i<n; i++) {
        a[i] = i;
        b[i] = i;
    }

    vectorAddSeq(n, a, b, result_s);
    vectorAddCuda(n, a, b, result);

    // verify the resuls
    for(int i=0; i<n; i++) {
        if (result[i]!=result_s[i]) {
            std::cout << "error in results! Element " << i << " is " << result[i] << ", but should be " << result_s[i] << std::endl;
            exit(1);
        }
    }

    std::cout << "results OK!" << std::endl;

    delete[] a;
    delete[] b;
    delete[] result;

    return 0;
}
