#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <iostream>
#include "timer.h"
#include <chrono>
using namespace std::chrono;

//using namespace std;

/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        std::cerr << "cuda error: " << hipGetErrorString(result) << std::endl;
        exit(1);
    }
}


__global__ void vectorTransformKernel(float* A, float* B, float* Result) {
// insert operation here


    for (int j=0; j<5; j++) {
        int i = threadIdx.x + blockDim.x * blockIdx.x;
        //Result[i] = Result[i] + A[i] * B[i];
        Result[i] += A[i] * B[i];


    }


//   sequentialTime.start();
//   for (j=0; j<5; j++) {
//     for (i=0; i<n; i++) {
// 	result[i] = result[i]+a[i]*b[i];
//     }
//   }
//   sequentialTime.stop();


}

void vectorTransformCuda(int n, float* a, float* b, float* result) {
    //int threadBlockSize = 512;

    int threadBlockSize;

    if (n < 512)
        threadBlockSize = n;
    else
        threadBlockSize = 512;


    // allocate the vectors on the GPU
    float* deviceA = NULL;
    checkCudaCall(hipMalloc((void **) &deviceA, n * sizeof(float)));
    if (deviceA == NULL) {
        std::cout << "could not allocate memory!" << std::endl;
        return;
    }
    float* deviceB = NULL;
    checkCudaCall(hipMalloc((void **) &deviceB, n * sizeof(float)));
    if (deviceB == NULL) {
        checkCudaCall(hipFree(deviceA));
        std::cout << "could not allocate memory!" << std::endl;
        return;
    }
    float* deviceResult = NULL;
    checkCudaCall(hipMalloc((void **) &deviceResult, n * sizeof(float)));
    if (deviceResult == NULL) {
        checkCudaCall(hipFree(deviceA));
        checkCudaCall(hipFree(deviceB));
        std::cout << "could not allocate memory!" << std::endl;
        return;
    }

    //timer kernelTime1 = timer("kernelTime1");
    //timer memoryTime = timer("memoryTime");

    high_resolution_clock::time_point t1 = high_resolution_clock::now();


    // copy the original vectors to the GPU
    // memoryTime.start();
    checkCudaCall(hipMemcpy(deviceA, a, n*sizeof(float), hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(deviceB, b, n*sizeof(float), hipMemcpyHostToDevice));
    // memoryTime.stop();

    high_resolution_clock::time_point t2 = high_resolution_clock::now();


    // execute kernel
    //kernelTime1.start();
    vectorTransformKernel<<<n/threadBlockSize, threadBlockSize>>>(deviceA, deviceB, deviceResult);
    hipDeviceSynchronize();
    //kernelTime1.stop();

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    // copy result back
    //memoryTime.start();
    high_resolution_clock::time_point t3 = high_resolution_clock::now();
    checkCudaCall(hipMemcpy(result, deviceResult, n * sizeof(float), hipMemcpyDeviceToHost));
    //memoryTime.stop();
    high_resolution_clock::time_point t4 = high_resolution_clock::now();


    checkCudaCall(hipFree(deviceA));
    checkCudaCall(hipFree(deviceB));
    checkCudaCall(hipFree(deviceResult));

    //std::cout << "vector-transform (kernel): \t\t" << kernelTime1  << std::endl;
    //std::cout << "vector-transform (memory): \t\t" << memoryTime << std::endl;

    std::cout << "vector-add (H2D):    \t\t" << duration_cast<microseconds>(t2 - t1).count() << "us" << std::endl;
    std::cout << "vector-add (kernel): \t\t" << duration_cast<microseconds>(t3 - t2).count() << "us" << std::endl;
    std::cout << "vector-add (D2H):    \t\t" << duration_cast<microseconds>(t4 - t3).count() << "us" << std::endl;

}

void vectorTransformSeq(int n, float* a, float* b, float* result) {
    int i,j; 

    //timer sequentialTime = timer("Sequential");
    high_resolution_clock::time_point t1 = high_resolution_clock::now();

  
    //sequentialTime.start();
    for (j=0; j<5; j++) {
        for (i=0; i<n; i++) {
            result[i] = result[i]+a[i]*b[i];
        }
    }
    //sequentialTime.stop();

    high_resolution_clock::time_point t2 = high_resolution_clock::now();

    std::cout << "vector-add (seq):    \t\t" << duration_cast<microseconds>(t2 - t1).count() << "us" << std::endl;

    //std::cout << "vector-transform (sequential): \t\t" << sequentialTime << std::endl;

}

int main(int argc, char* argv[]) {
    //int n = 655360;
    int n = atoi(getenv("VECTOR_ADD_N"));
    float* a = new float[n];
    float* b = new float[n];
    float* result = new float[n];
    float* result_s = new float[n];

    if (argc > 1) n = atoi(argv[1]);

    std::cout << "Iteratively transform vector A with vector B of " << n << " integer elements." << std::endl;
    // initialize the vectors.
    for(int i=0; i<n; i++) {
        a[i] = i;
        b[i] = 0.1*i;
	    result[i]=0;
	    result_s[i]=0;
    }

    vectorTransformSeq(n, a, b, result_s);
    vectorTransformCuda(n, a, b, result);
    
    // verify the resuls
    for(int i=0; i<n; i++) {
        //if (result[i]!=result_s[i]) {
        if (fabs(result[i] - result_s[i]) >0.001) {
            std::cout << "error in results! Element " << i << " is " << std::to_string(result[i]) << ", but should be " << std::to_string(result_s[i]) << std::endl; 
            exit(1);
        }
    }
    std::cout << "results OK!" << std::endl;
            
    delete[] a;
    delete[] b;
    delete[] result;
    
    return 0;
}
